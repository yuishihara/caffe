#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void RMSPropGravesUpdate(int N, Dtype* grad, Dtype* n, Dtype* g, Dtype* delta,
    Dtype rms_decay, Dtype momentum, Dtype epsilon, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float gradi = grad[i];
    float ni = n[i] = rms_decay * n[i] + (1 - rms_decay) * gradi * gradi;
    float gi = g[i] = rms_decay * g[i] + (1 - rms_decay) * gradi;
    float di = delta[i] = momentum * delta[i] -
        local_rate * gradi / sqrt(ni - gi * gi + epsilon);
    grad[i] = -di;
  }
}
template <typename Dtype>
void rmspropgraves_update_gpu(int N, Dtype* grad, Dtype* n, Dtype* g, Dtype* delta,
    Dtype rms_decay, Dtype momentum, Dtype epsilon, Dtype local_rate) {
  RMSPropGravesUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, grad, n, g, delta, rms_decay, momentum, epsilon, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void rmspropgraves_update_gpu<float>(int, float*, float*, float*, float*,
    float, float, float, float);
template void rmspropgraves_update_gpu<double>(int, double*, double*, double*, double*,
    double, double, double, double);

}  // namespace caffe
