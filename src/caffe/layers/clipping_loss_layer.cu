#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/clipping_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Clip(int N, Dtype* clipped) {
  CUDA_KERNEL_LOOP(i, N) {
    if (Dtype(1) < clipped[i]) {
      clipped[i] = Dtype(1);
    } else if (clipped[i] < Dtype(-1)) {
      clipped[i] = Dtype(-1);
    }
  }
}

template <typename Dtype>
void ClippingLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  if (Dtype(1) <= dot) {
    dot = Dtype(1);
  }
  Dtype loss = dot;
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void ClippingLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0];

      const int N = bottom[i]->count();
      Clip<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
                  N, diff_.mutable_gpu_data());
      CUDA_POST_KERNEL_CHECK;

      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ClippingLossLayer);

}  // namespace caffe
